
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void device_saxpy(int n, float a, float * x, float * y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

void host_saxpy(int n, float a, float* x, float* y)
{
	for(int i=0;i<n;i++) y[i] = a*x[i] + y[i];
}

int main()
{
	int ARRAY_SIZE = 1<<10;

	// device saxpy arrays
	float * x1; 
	float * y1;
	float * d_x1;
	float * d_y1;

	// host saxpy arrays
	float * x2;
	float * y2;

	// Allocate host arrays
	x1 = (float*)malloc(sizeof(float) * ARRAY_SIZE);
	y1 = (float*)malloc(sizeof(float) * ARRAY_SIZE);
	x2 = (float*)malloc(sizeof(float) * ARRAY_SIZE);
	y2 = (float*)malloc(sizeof(float) * ARRAY_SIZE);

	// Initialize arrays
	for (int i=0; i<ARRAY_SIZE; i++)
	{
		x1[i] = (float)i+100.0;
		y1[i] = (float)-i;

		x2[i] = (float)i+100.0;
		y2[i] = (float)-i;
  	}

    	// Allocate device memory for x,y
    	hipMalloc((void**)&d_x1, sizeof(float) * ARRAY_SIZE);
    	hipMalloc((void**)&d_y1, sizeof(float) * ARRAY_SIZE);

	// Copy the device arrays
	hipMemcpy(d_x1, x1, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y1, y1, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

	// Perform SAXPY in the device
	printf("Computing SAXPY on the GPU...");
	device_saxpy<<<(ARRAY_SIZE+255)/256,256>>>(ARRAY_SIZE, 2.0, d_x1, d_y1);
	hipDeviceSynchronize();
	printf("Done\n");

	// Copy back the results from the device
	hipMemcpy(x1, d_x1, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y1, d_y1, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);

	// Perform SAXPY in the host 
	printf("Computing SAXPY on the CPU...");
	host_saxpy(ARRAY_SIZE,2.0,x2,y2);
	printf("Done\n");

	// Compare the results
	printf("Comparing the output for each implementation...");
	float max_error = 0.0001f;
	bool correct = true;
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		if(abs(y1[i]-y2[i])>max_error)
			correct = false;
	}
	if(correct)
		printf("Correct :)\n");
	else
		printf("Incorrect :(\n");
			
	// Cleanup after executions
	hipFree(d_x1);
	hipFree(d_y1);
	free(x1);
	free(y1);
}

