
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello()
{
	printf("Hello World! My thread ID is %d \n",threadIdx.x);
}

int main() 
{
	cuda_hello<<<1,256>>>();
	hipDeviceSynchronize();
	return 0;
}
